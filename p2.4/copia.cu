#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

__global__ void RGBtoGrayscale(unsigned char *image, int width, int height, unsigned char *grayscale) {
//rellenar
	//Ancho de la imagen / los threads y que vayan cambiando la imagen de manera vertical
        int col = blockIdx.x*blockDim.x+threadIdx.x;
        int fila = blockIdx.y *blockDim.y + threadIdx.y;
	if(col >= width || fila >= height){ return;}
	int idx = fila * width * 3 + col * 3;
        float pixel = 0.299f * image[idx]+ 0.587f * image[idx+1] + 0.114f * image[idx+2];
        grayscale[idx] =(unsigned char) pixel;
        grayscale[idx+1] = (unsigned char) pixel;
        grayscale[idx+2] =(unsigned char)  pixel;

}

#define NUM_CHANNELS 3

int main(int argc, char **argv) {
  if (argc != 2) {
    printf("Uso: %s <nombre_imagen>\n", argv[0]);
    return -1;
  }

  // Load image using OpenCV
  cv::Mat image = cv::imread(argv[1], cv::IMREAD_COLOR);
  int width = image.cols;
  int height = image.rows;

  printf("cols: %d rows: %d\n", image.cols, image.rows);

  // cv::imshow("Original", image);
  // cv::waitKey(0);

  // Allocate memory for grayscale image on host and device
  unsigned char *host_image = (unsigned char*)image.data;
  unsigned char *device_image, *device_grayscale;
  hipMalloc(&device_image, width * height * NUM_CHANNELS * sizeof(unsigned char));
  hipMalloc(&device_grayscale, width * height * NUM_CHANNELS * sizeof(unsigned char));

  // Copy image to device memory
  hipMemcpy(device_image, host_image, width * height * NUM_CHANNELS * sizeof(unsigned char), hipMemcpyHostToDevice);

  // Define thread block size and grid size
  int blockLen = 16; 
  dim3 dimGrid((width/blockLen)+1, (height/blockLen)+1);
  dim3 dimBlocks(blockLen, blockLen);
  printf("dimGrid: %d cols (width), %d rows (height)\n", dimGrid.x, dimGrid.y);

  // Launch kernel to convert to grayscale
  RGBtoGrayscale<<<dimGrid, dimBlocks>>>(device_image, width, height, device_grayscale);
  assert(hipDeviceSynchronize() == 0);

  // Copy grayscale image back to host memory
  hipMemcpy(host_image, device_grayscale, width * height * NUM_CHANNELS * sizeof(unsigned char), hipMemcpyDeviceToHost);

  // Show grayscale image
  cv::imshow("Grayscale Image", image);
  cv::waitKey(0);
  cv::imwrite("imagen_gris.png", image);

  // Free memory
  hipFree(device_image);
  hipFree(device_grayscale);

  return 0;
}

