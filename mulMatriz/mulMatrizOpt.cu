//------------------------------------------------------------------+
// PCM. Arquitecturas Paralelas Curso 20/21 ETSISI        22/04/21  |
//                                                                  |
// mulmatrizOpt.c: Prueba de multiplicacion de matriz x matriz en la|
//                 CPU para luego comparar con version GPU          |
//                    VERSION OPTIMIZADA [Acceso a memoria]         |
//------------------------------------------------------------------+


#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define anchoBloque 16  // En 2D => 16x16 = 256 threads
//-------------------------------------------------------------------
void initMatriz (double *M, int card, double valor) {
  int i;
  for (i=0; i<card; i++) {
    M[i] = valor;
    valor += 0.1;
  }
}

//-------------------------------------------------------------------
__global__ void mulMatrizKernel (double *Ad, double *Bd, double *Cd, int card) {
  __shared__ double Ads[anchoBloque][anchoBloque];
  __shared__ double Bds[anchoBloque][anchoBloque];
  int tx  = threadIdx.x;
  int ty  = threadIdx.y;
  int fil = blockIdx.y*anchoBloque+ty;
  int col = blockIdx.x*anchoBloque+tx;
  int k,m;
  double Cvalor = 0.0;

  for (m=0; m<card/anchoBloque; m++) {
    Ads[ty][tx] = Ad[fil*card + (m*anchoBloque + tx)];
    Bds[ty][tx] = Bd[(m*anchoBloque + ty)*card + col];
    __syncthreads();
    for (k=0; k<anchoBloque; k++)
      Cvalor += Ads[ty][k] * Bds[k][tx];
    __syncthreads();
  }
  Cd[fil * card + col] = Cvalor;
}

//-------------------------------------------------------------------
int main (int argc, char *argv[])
{
  int filA, colA, filB, colB, filC, colC;
  struct timeval t0, tf, t;
  double *A, *B, *C;
  double *Ad, *Bd, *Cd;
  int    sizeA, sizeB, sizeC, f, k;
  double valor;

  filA = atoi(argv[1]);
  colA = filA;
  filB = filA;
  colB = filA;
  filC = filA;
  colC = filA;
  sizeA = filA*colA*sizeof(double);
  sizeB = filB*colB*sizeof(double);
  sizeC = filC*colC*sizeof(double);
  A = (double *) malloc (sizeA);
  B = (double *) malloc (sizeB);
  C = (double *) malloc (sizeC);
  initMatriz (A, filA*colA, 1.0f );
  initMatriz (B, filB*colB, 0.01f);

  assert (gettimeofday (&t0, NULL) == 0);
  // Transferir A y B a la GPU
  hipMalloc ((void**) &Ad, sizeA);
  hipMemcpy (Ad, A, sizeA, hipMemcpyHostToDevice);
  hipMalloc ((void**) &Bd, sizeB);
  hipMemcpy (Bd, B, sizeB, hipMemcpyHostToDevice);
  // Ubicar C en la CPU
  hipMalloc ((void**) &Cd, sizeC);
  // Invocar al kernel
  dim3 dimGrid (filA/anchoBloque, filA/anchoBloque);
  dim3 dimBlock(anchoBloque, anchoBloque);
  mulMatrizKernel<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, filA);
  hipDeviceSynchronize();
  // Transferir C desde la GPU
  hipMemcpy (C, Cd, sizeC, hipMemcpyDeviceToHost);
  // Liberar matrices en la GPU
  hipFree(Ad); hipFree(Bd); hipFree(Cd);
  assert (gettimeofday (&tf, NULL) == 0);

  timersub (&tf, &t0, &t);
  printf ("Tiempo = %ld:%ld \n", t.tv_sec, t.tv_usec);

  // Comprobamos diagonal principal
  for (f=0; f<filA; f++) {
    valor = 0.0;
    for (k=0; k<filA; k++)
      valor += A[f * colA + k] * B[k * colA + f];
    if (fabs(valor - C[f*colA+f]) > 0.1) {
      printf ("Error f=%d c=%d CPU = %lf GPU = %lf diff = %lf\n",
      f, f, valor, C[f*colA+f], valor-C[f*colA+f]);
      return 0;
    }
  }
  return 0;
}
